// Hello world, CUDA version

// moveArrays.cu 
// 
// demonstrates CUDA interface to data allocation on device (GPU) 
// and data movement between host (CPU) and device.
#include <stdio.h> 
#include <assert.h> 
#include <hip/hip_runtime.h> 

int main(void) { 
    float *a_h, *b_h; // pointers to host memory 
    float *a_d, *b_d; // pointers to device memory 
    int N = 14; 
    int i; 
    
    // allocate arrays on host 
    a_h = (float *)malloc(sizeof(float)*N);
    b_h = (float *)malloc(sizeof(float)*N);

    // allocate arrays on device 
    hipMalloc((void **) &a_d, sizeof(float)*N);
    hipMalloc((void **) &b_d, sizeof(float)*N);

    // initialize host data 
    for (i=0; i<N; i++) { 
        a_h[i] = 10.f+i; 
        b_h[i] = 0.f; 
    } 
    
    // send data from host to device: a_h to a_d 
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    // copy data within device: a_d to b_d 
    hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
    // retrieve data from device: b_d to b_h 
    hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    // check result 
    for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);
    // cleanup 
    
    free(a_h);
    free(b_h);
    hipFree(a_d);
    hipFree(b_d);

    return 0;
}

